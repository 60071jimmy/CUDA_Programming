/*	Basic Calculation Use CUDA，使用CUDA執行基本運算	*/

//-----include header files, 引入標頭檔-----
#include "hip/hip_runtime.h"												//	include "cuda_runtime.h" header file, 引入標頭檔cuda_runtime.h
									//	include "device_launch_parameters.h" header file, 引入標頭檔device_launch_parameters.h

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

