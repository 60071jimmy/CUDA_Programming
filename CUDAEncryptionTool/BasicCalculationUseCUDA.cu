/*	Basic Calculation Use CUDA�A�ϥ�CUDA����򥻹B��	*/

//-----include header files, �ޤJ���Y��-----
#include "hip/hip_runtime.h"												//	include "cuda_runtime.h" header file, �ޤJ���Y��cuda_runtime.h
									//	include "device_launch_parameters.h" header file, �ޤJ���Y��device_launch_parameters.h

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

