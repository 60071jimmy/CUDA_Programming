/*	
	Develop by Jimmy Hu, 2017
*/
//-----include header files, �ޤJ���Y��-----
//	-----include CUDA libraries-----
#include "Libraries.cuh";												//	include "Libraries.cuh" header file, �ޤJ���Y��Libraries.cuh
//	-----include C standard libraries
#include <stdio.h>														//	include "stdio.h" header file, �ޤJ���Y��stdio.h
#include <stdlib.h>														//	include "stdlib.h" header file, �ޤJ���Y��stdlib.h

int main()																//	main function, �D�{��
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    //***Add vectors in parallel, ����p��}�C�[�k***
	hipError_t cudaStatus;												//	�ŧicudaStatus����A�Ω�O�����浲�G�O�_���~
	cudaStatus = hipDeviceReset();										//	���mCUDA�p��˸m
	if (cudaStatus != hipSuccess) {									//	�YcudaStatus��hipSuccess
		fprintf(stderr, "hipDeviceReset failed!");						//	��ܿ��~�T��
		return 1;														//	�^��1�õ����{��
	}																	//	����if�ԭz

	cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	system("pause");
    return 0;
}
